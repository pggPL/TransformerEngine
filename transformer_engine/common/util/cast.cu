/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_runtime.h>
#include <transformer_engine/cast.h>

#include <cfloat>
#include <limits>

#include "../common.h"
#include "../util/vectorized_pointwise.h"
#include "../utils.cuh"
#include "cast_kernels.cuh"
#include "math.h"
#include "ptx.cuh"
#include "transformer_engine/transpose.h"
#include "transformer_engine/activation.h"
#include "../transpose/cast_transpose.h"


void nvte_quantize(const NVTETensor input, NVTETensor output, hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = false;
  constexpr bool IS_DACT = false;
  constexpr bool IS_ACT = false;
  constexpr NVTETensor dbias = nullptr;
  constexpr NVTETensor workspace = nullptr;
  constexpr const NVTETensor activation_input = nullptr;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, nullptr>
      (input, activation_input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_noop(const NVTETensor input, NVTETensor output, NVTETensor noop, hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_noop);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = false;
  constexpr bool IS_DACT = false;
  constexpr bool IS_ACT = false;
  constexpr NVTETensor dbias = nullptr;
  constexpr NVTETensor workspace = nullptr;
  constexpr const NVTETensor activation_input = nullptr;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, nullptr>
      (input, activation_input, noop, output, dbias, workspace, stream);
}

void nvte_quantize_dbias(const NVTETensor input, NVTETensor output, NVTETensor dbias,
                             NVTETensor workspace, hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = false;
  constexpr bool IS_ACT = false;
  constexpr const NVTETensor activation_input = nullptr;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, nullptr>
      (input, activation_input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_dgelu(const NVTETensor input, const NVTETensor activation_input,
                                   NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                                   hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_dgelu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, dgelu<fp32, fp32>>
      (input, activation_input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_dsilu(const NVTETensor input, const NVTETensor activation_input,
                                   NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                                   hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_dsilu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, dsilu<fp32, fp32>>
      (input, activation_input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_drelu(const NVTETensor input, const NVTETensor activation_input,
                                   NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                                   hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_drelu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, drelu<fp32, fp32>>
      (input, activation_input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_dqgelu(const NVTETensor input, const NVTETensor activation_input,
                                    NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                                    hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_dqgelu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, dqgelu<fp32, fp32>>
      (input, activation_input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_dsrelu(const NVTETensor input, const NVTETensor activation_input,
                                    NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                                    hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_dsrelu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, dsrelu<fp32, fp32>>
      (input, activation_input, nullptr, output, dbias, workspace, stream);
}

void nvte_dequantize(const NVTETensor input, NVTETensor output, hipStream_t stream) {
  NVTE_API_CALL(nvte_dequantize);
  using namespace transformer_engine;
  fp8_dequantize(*reinterpret_cast<const Tensor *>(input), reinterpret_cast<Tensor *>(output),
                 stream);
}
