/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"

at::Tensor cast_to_fp8(const at::Tensor& input, const at::Tensor& scale, at::Tensor amax,
                       at::Tensor scale_inv, transformer_engine::DType otype,
                       std::vector<int64_t> scaling_mode, const int scale_offset,
                       const int amax_offset, const int scale_inv_offset) {
  using namespace transformer_engine;
  auto input_shape = input.sizes().vec();
  std::vector<size_t> shape{input_shape.begin(), input_shape.end()};

  auto output = at::empty_like(input, at::CUDA(GetATenDType(otype)));

  if (input.numel() == 0) return output;

  // Get pointers for FP8 scale, amax, scale-inverse
  void* scale_dptr = getDataPtr(scale, scale_offset);
  void* amax_dptr = getDataPtr(amax, amax_offset);
  void* scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  NVTEScalingMode nvte_scaling_mode = {scaling_mode[0], scaling_mode[1], scaling_mode[2]};

  auto input_cu = makeTransformerEngineTensor(input);
  auto output_cu =
      makeTransformerEngineTensor(output.data_ptr(), shape, otype, amax_dptr, scale_dptr,
                                  scale_inv_dptr, getTensorShape(scale_inv), nvte_scaling_mode);

  nvte_fp8_quantize(input_cu.data(), output_cu.data(), at::cuda::getCurrentCUDAStream());

  return output;
}

void cast_to_fp8_noalloc(const at::Tensor& input, const at::Tensor& scale, at::Tensor output,
                         at::Tensor amax, at::Tensor scale_inv, transformer_engine::DType otype,
                         std::vector<int64_t> scaling_mode, const int scale_offset,
                         const int amax_offset, const int scale_inv_offset) {
  using namespace transformer_engine;
  auto input_shape = input.sizes().vec();
  std::vector<size_t> shape{input_shape.begin(), input_shape.end()};

  // Get pointers for FP8 scale, amax, scale-inverse
  void* scale_dptr = getDataPtr(scale, scale_offset);
  void* amax_dptr = getDataPtr(amax, amax_offset);
  void* scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  NVTEScalingMode nvte_scaling_mode = {scaling_mode[0], scaling_mode[1], scaling_mode[2]};

  auto input_cu = makeTransformerEngineTensor(input);
  auto output_cu =
      makeTransformerEngineTensor(output.data_ptr(), shape, otype, amax_dptr, scale_dptr,
                                  scale_inv_dptr, getTensorShape(scale_inv), nvte_scaling_mode);

  nvte_fp8_quantize(input_cu.data(), output_cu.data(), at::cuda::getCurrentCUDAStream());

  return;
}

at::Tensor cast_from_fp8(const at::Tensor& input, const at::Tensor& scale_inv,
                         transformer_engine::DType itype, transformer_engine::DType otype,
                         const int scale_inv_offset) {
  using namespace transformer_engine;
  auto input_shape = input.sizes().vec();
  std::vector<size_t> shape{input_shape.begin(), input_shape.end()};

  auto output = at::empty_like(input, at::CUDA(GetATenDType(otype)));

  auto input_cu = makeTransformerEngineTensor(input.data_ptr(), shape, itype, nullptr, nullptr,
                                              getDataPtr(scale_inv, scale_inv_offset));
  auto output_cu = makeTransformerEngineTensor(output);

  nvte_fp8_dequantize(input_cu.data(), output_cu.data(), at::cuda::getCurrentCUDAStream());

  return output;
}

std::vector<at::Tensor> fp8_cast_dbias(const at::Tensor& input, const at::Tensor& scale,
                                       at::Tensor amax, at::Tensor scale_inv,
                                       transformer_engine::DType otype,
                                       std::vector<int64_t> scaling_mode, const int scale_offset,
                                       const int amax_offset, const int scale_inv_offset) {
  using namespace transformer_engine;
  auto input_shape = input.sizes().vec();
  std::vector<size_t> shape{input_shape.begin(), input_shape.end()};

  DType grad_output_type = GetTransformerEngineDType(input.scalar_type());
  auto output = at::empty_like(input, at::CUDA(GetATenDType(otype)));
  auto grad_bias = allocateTorchTensor(input.size(-1), grad_output_type);

  if (input.numel() == 0) return {grad_bias, output};

  // Get pointers for FP8 scale, amax, scale-inverse
  void* scale_dptr = getDataPtr(scale, scale_offset);
  void* amax_dptr = getDataPtr(amax, amax_offset);
  void* scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  NVTEScalingMode nvte_scaling_mode = {scaling_mode[0], scaling_mode[1], scaling_mode[2]};

  auto input_cu = makeTransformerEngineTensor(input);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);
  auto output_cu =
      makeTransformerEngineTensor(output.data_ptr(), shape, otype, amax_dptr, scale_dptr,
                                  scale_inv_dptr, getTensorShape(scale_inv), nvte_scaling_mode);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias(input_cu.data(), output_cu.data(), dbias_cu.data(), workspace.data(),
                          at::cuda::getCurrentCUDAStream());

  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  nvte_fp8_quantize_dbias(input_cu.data(), output_cu.data(), dbias_cu.data(), workspace.data(),
                          at::cuda::getCurrentCUDAStream());

  return {grad_bias, output};
}

std::vector<at::Tensor> fp8_cast_dbias_dgelu(at::Tensor grad_output, at::Tensor act_input,
                                             at::Tensor scale, at::Tensor amax,
                                             at::Tensor scale_inv, transformer_engine::DType otype,
                                             std::vector<int64_t> scaling_mode, int scale_offset,
                                             int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* scale_dptr = getDataPtr(scale, scale_offset);
  void* amax_dptr = getDataPtr(amax, amax_offset);
  void* scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  NVTEScalingMode nvte_scaling_mode = {scaling_mode[0], scaling_mode[1], scaling_mode[2]};

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto cast_output_cu =
      makeTransformerEngineTensor(dact.data_ptr(), {M, N}, otype, amax_dptr, scale_dptr,
                                  scale_inv_dptr, getTensorShape(scale_inv), nvte_scaling_mode);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_dgelu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                dbias_cu.data(), workspace.data(),
                                at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_dgelu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                dbias_cu.data(), workspace.data(),
                                at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact};
}

std::vector<at::Tensor> fp8_cast_dbias_dsilu(at::Tensor grad_output, at::Tensor act_input,
                                             at::Tensor scale, at::Tensor amax,
                                             at::Tensor scale_inv, transformer_engine::DType otype,
                                             std::vector<int64_t> scaling_mode, int scale_offset,
                                             int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* scale_dptr = getDataPtr(scale, scale_offset);
  void* amax_dptr = getDataPtr(amax, amax_offset);
  void* scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  NVTEScalingMode nvte_scaling_mode = {scaling_mode[0], scaling_mode[1], scaling_mode[2]};

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto cast_output_cu =
      makeTransformerEngineTensor(dact.data_ptr(), {M, N}, otype, amax_dptr, scale_dptr,
                                  scale_inv_dptr, getTensorShape(scale_inv), nvte_scaling_mode);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_dsilu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                dbias_cu.data(), workspace.data(),
                                at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_dsilu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                dbias_cu.data(), workspace.data(),
                                at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact};
}

std::vector<at::Tensor> fp8_cast_dbias_drelu(at::Tensor grad_output, at::Tensor act_input,
                                             at::Tensor scale, at::Tensor amax,
                                             at::Tensor scale_inv, transformer_engine::DType otype,
                                             std::vector<int64_t> scaling_mode, int scale_offset,
                                             int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* scale_dptr = getDataPtr(scale, scale_offset);
  void* amax_dptr = getDataPtr(amax, amax_offset);
  void* scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  NVTEScalingMode nvte_scaling_mode = {scaling_mode[0], scaling_mode[1], scaling_mode[2]};

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto cast_output_cu =
      makeTransformerEngineTensor(dact.data_ptr(), {M, N}, otype, amax_dptr, scale_dptr,
                                  scale_inv_dptr, getTensorShape(scale_inv), nvte_scaling_mode);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_drelu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                dbias_cu.data(), workspace.data(),
                                at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_drelu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                dbias_cu.data(), workspace.data(),
                                at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact};
}

std::vector<at::Tensor> fp8_cast_dbias_dqgelu(at::Tensor grad_output, at::Tensor act_input,
                                              at::Tensor scale, at::Tensor amax,
                                              at::Tensor scale_inv, transformer_engine::DType otype,
                                              std::vector<int64_t> scaling_mode, int scale_offset,
                                              int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* scale_dptr = getDataPtr(scale, scale_offset);
  void* amax_dptr = getDataPtr(amax, amax_offset);
  void* scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  NVTEScalingMode nvte_scaling_mode = {scaling_mode[0], scaling_mode[1], scaling_mode[2]};

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto cast_output_cu =
      makeTransformerEngineTensor(dact.data_ptr(), {M, N}, otype, amax_dptr, scale_dptr,
                                  scale_inv_dptr, getTensorShape(scale_inv), nvte_scaling_mode);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_dqgelu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                 dbias_cu.data(), workspace.data(),
                                 at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_dqgelu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                 dbias_cu.data(), workspace.data(),
                                 at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact};
}

std::vector<at::Tensor> fp8_cast_dbias_dsrelu(at::Tensor grad_output, at::Tensor act_input,
                                              at::Tensor scale, at::Tensor amax,
                                              at::Tensor scale_inv, transformer_engine::DType otype,
                                              std::vector<int64_t> scaling_mode, int scale_offset,
                                              int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* scale_dptr = getDataPtr(scale, scale_offset);
  void* amax_dptr = getDataPtr(amax, amax_offset);
  void* scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  NVTEScalingMode nvte_scaling_mode = {scaling_mode[0], scaling_mode[1], scaling_mode[2]};

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto cast_output_cu =
      makeTransformerEngineTensor(dact.data_ptr(), {M, N}, otype, amax_dptr, scale_dptr,
                                  scale_inv_dptr, getTensorShape(scale_inv), nvte_scaling_mode);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_dsrelu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                 dbias_cu.data(), workspace.data(),
                                 at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_dsrelu(input_cu.data(), act_input_cu.data(), cast_output_cu.data(),
                                 dbias_cu.data(), workspace.data(),
                                 at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact};
}

std::vector<at::Tensor> fp8_cast_dbias_x2(const at::Tensor& input, const at::Tensor& scale,
                                          at::Tensor amax, at::Tensor scale_inv,
                                          transformer_engine::DType otype, const int scale_offset,
                                          const int amax_offset, const int scale_inv_offset) {
  using namespace transformer_engine;
  auto input_shape = input.sizes().vec();
  std::vector<size_t> shape{input_shape.begin(), input_shape.end()};

  DType grad_output_type = GetTransformerEngineDType(input.scalar_type());
  auto output_rowwise = at::empty_like(input, at::CUDA(GetATenDType(otype)));
  auto output_columnwise = at::empty_like(input, at::CUDA(GetATenDType(otype)));
  auto grad_bias = allocateTorchTensor(input.size(-1), grad_output_type);

  if (input.numel() == 0) return {grad_bias, output_rowwise, output_columnwise};

  // Get pointers for FP8 scale, amax, scale-inverse
  void* rowwise_scale_dptr = getDataPtr(scale, scale_offset);
  void* rowwise_amax_dptr = getDataPtr(amax, amax_offset);
  void* rowwise_scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  auto columnwise_scale = scale.detach().clone();
  auto columnwise_scale_inv = scale_inv.detach().clone();
  auto columnwise_amax = amax.detach().clone();
  void* columnwise_scale_dptr = getDataPtr(columnwise_scale, scale_offset);
  void* columnwise_amax_dptr = getDataPtr(columnwise_amax, amax_offset);
  void* columnwise_scale_inv_dptr = getDataPtr(columnwise_scale_inv, scale_inv_offset);

  auto input_cu = makeTransformerEngineTensor(input);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);
  auto rowwise_output_cu =
      makeTransformerEngineTensor(output_rowwise.data_ptr(), shape, otype, rowwise_amax_dptr,
                                  rowwise_scale_dptr, rowwise_scale_inv_dptr);
  auto columnwise_output_cu =
      makeTransformerEngineTensor(output_columnwise.data_ptr(), shape, otype, columnwise_amax_dptr,
                                  columnwise_scale_dptr, columnwise_scale_inv_dptr);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_x2(input_cu.data(), rowwise_output_cu.data(), columnwise_output_cu.data(),
                             dbias_cu.data(), workspace.data(), at::cuda::getCurrentCUDAStream());

  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  nvte_fp8_quantize_dbias_x2(input_cu.data(), rowwise_output_cu.data(), columnwise_output_cu.data(),
                             dbias_cu.data(), workspace.data(), at::cuda::getCurrentCUDAStream());

  return {grad_bias, output_rowwise, output_columnwise};
}

std::vector<at::Tensor> fp8_cast_dbias_dgelu_x2(at::Tensor grad_output, at::Tensor act_input,
                                                at::Tensor scale, at::Tensor amax,
                                                at::Tensor scale_inv,
                                                transformer_engine::DType otype, int scale_offset,
                                                int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* rowwise_scale_dptr = getDataPtr(scale, scale_offset);
  void* rowwise_amax_dptr = getDataPtr(amax, amax_offset);
  void* rowwise_scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  auto columnwise_scale = scale.detach().clone();
  auto columnwise_scale_inv = scale_inv.detach().clone();
  auto columnwise_amax = amax.detach().clone();
  void* columnwise_scale_dptr = getDataPtr(columnwise_scale, scale_offset);
  void* columnwise_amax_dptr = getDataPtr(columnwise_amax, amax_offset);
  void* columnwise_scale_inv_dptr = getDataPtr(columnwise_scale_inv, scale_inv_offset);

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact_rowwise = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto dact_columnwise =
      allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto rowwise_output_cu =
      makeTransformerEngineTensor(dact_rowwise.data_ptr(), {M, N}, otype, rowwise_amax_dptr,
                                  rowwise_scale_dptr, rowwise_scale_inv_dptr);
  auto columnwise_output_cu =
      makeTransformerEngineTensor(dact_columnwise.data_ptr(), {M, N}, otype, columnwise_amax_dptr,
                                  columnwise_scale_dptr, columnwise_scale_inv_dptr);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_dgelu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                   columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                   at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_dgelu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                   columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                   at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact_rowwise, dact_columnwise};
}

std::vector<at::Tensor> fp8_cast_dbias_dsilu_x2(at::Tensor grad_output, at::Tensor act_input,
                                                at::Tensor scale, at::Tensor amax,
                                                at::Tensor scale_inv,
                                                transformer_engine::DType otype, int scale_offset,
                                                int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* rowwise_scale_dptr = getDataPtr(scale, scale_offset);
  void* rowwise_amax_dptr = getDataPtr(amax, amax_offset);
  void* rowwise_scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  auto columnwise_scale = scale.detach().clone();
  auto columnwise_scale_inv = scale_inv.detach().clone();
  auto columnwise_amax = amax.detach().clone();
  void* columnwise_scale_dptr = getDataPtr(columnwise_scale, scale_offset);
  void* columnwise_amax_dptr = getDataPtr(columnwise_amax, amax_offset);
  void* columnwise_scale_inv_dptr = getDataPtr(columnwise_scale_inv, scale_inv_offset);

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact_rowwise = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto dact_columnwise =
      allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto rowwise_output_cu =
      makeTransformerEngineTensor(dact_rowwise.data_ptr(), {M, N}, otype, rowwise_amax_dptr,
                                  rowwise_scale_dptr, rowwise_scale_inv_dptr);
  auto columnwise_output_cu =
      makeTransformerEngineTensor(dact_columnwise.data_ptr(), {M, N}, otype, columnwise_amax_dptr,
                                  columnwise_scale_dptr, columnwise_scale_inv_dptr);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_dsilu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                   columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                   at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_dsilu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                   columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                   at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact_rowwise, dact_columnwise};
}

std::vector<at::Tensor> fp8_cast_dbias_drelu_x2(at::Tensor grad_output, at::Tensor act_input,
                                                at::Tensor scale, at::Tensor amax,
                                                at::Tensor scale_inv,
                                                transformer_engine::DType otype, int scale_offset,
                                                int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* rowwise_scale_dptr = getDataPtr(scale, scale_offset);
  void* rowwise_amax_dptr = getDataPtr(amax, amax_offset);
  void* rowwise_scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  auto columnwise_scale = scale.detach().clone();
  auto columnwise_scale_inv = scale_inv.detach().clone();
  auto columnwise_amax = amax.detach().clone();
  void* columnwise_scale_dptr = getDataPtr(columnwise_scale, scale_offset);
  void* columnwise_amax_dptr = getDataPtr(columnwise_amax, amax_offset);
  void* columnwise_scale_inv_dptr = getDataPtr(columnwise_scale_inv, scale_inv_offset);

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact_rowwise = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto dact_columnwise =
      allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto rowwise_output_cu =
      makeTransformerEngineTensor(dact_rowwise.data_ptr(), {M, N}, otype, rowwise_amax_dptr,
                                  rowwise_scale_dptr, rowwise_scale_inv_dptr);
  auto columnwise_output_cu =
      makeTransformerEngineTensor(dact_columnwise.data_ptr(), {M, N}, otype, columnwise_amax_dptr,
                                  columnwise_scale_dptr, columnwise_scale_inv_dptr);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_drelu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                   columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                   at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_drelu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                   columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                   at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact_rowwise, dact_columnwise};
}

std::vector<at::Tensor> fp8_cast_dbias_dqgelu_x2(at::Tensor grad_output, at::Tensor act_input,
                                                 at::Tensor scale, at::Tensor amax,
                                                 at::Tensor scale_inv,
                                                 transformer_engine::DType otype, int scale_offset,
                                                 int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* rowwise_scale_dptr = getDataPtr(scale, scale_offset);
  void* rowwise_amax_dptr = getDataPtr(amax, amax_offset);
  void* rowwise_scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  auto columnwise_scale = scale.detach().clone();
  auto columnwise_scale_inv = scale_inv.detach().clone();
  auto columnwise_amax = amax.detach().clone();
  void* columnwise_scale_dptr = getDataPtr(columnwise_scale, scale_offset);
  void* columnwise_amax_dptr = getDataPtr(columnwise_amax, amax_offset);
  void* columnwise_scale_inv_dptr = getDataPtr(columnwise_scale_inv, scale_inv_offset);

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact_rowwise = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto dact_columnwise =
      allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto rowwise_output_cu =
      makeTransformerEngineTensor(dact_rowwise.data_ptr(), {M, N}, otype, rowwise_amax_dptr,
                                  rowwise_scale_dptr, rowwise_scale_inv_dptr);
  auto columnwise_output_cu =
      makeTransformerEngineTensor(dact_columnwise.data_ptr(), {M, N}, otype, columnwise_amax_dptr,
                                  columnwise_scale_dptr, columnwise_scale_inv_dptr);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_dqgelu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                    columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                    at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_dqgelu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                    columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                    at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact_rowwise, dact_columnwise};
}

std::vector<at::Tensor> fp8_cast_dbias_dsrelu_x2(at::Tensor grad_output, at::Tensor act_input,
                                                 at::Tensor scale, at::Tensor amax,
                                                 at::Tensor scale_inv,
                                                 transformer_engine::DType otype, int scale_offset,
                                                 int amax_offset, int scale_inv_offset) {
  using namespace transformer_engine;

  // Tensor dimensions
  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  // Get pointers for FP8 scale, amax, scale-inverse
  void* rowwise_scale_dptr = getDataPtr(scale, scale_offset);
  void* rowwise_amax_dptr = getDataPtr(amax, amax_offset);
  void* rowwise_scale_inv_dptr = getDataPtr(scale_inv, scale_inv_offset);
  auto columnwise_scale = scale.detach().clone();
  auto columnwise_scale_inv = scale_inv.detach().clone();
  auto columnwise_amax = amax.detach().clone();
  void* columnwise_scale_dptr = getDataPtr(columnwise_scale, scale_offset);
  void* columnwise_amax_dptr = getDataPtr(columnwise_amax, amax_offset);
  void* columnwise_scale_inv_dptr = getDataPtr(columnwise_scale_inv, scale_inv_offset);

  // Construct Transformer Engine tensors
  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dact_rowwise = allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto dact_columnwise =
      allocateTorchTensor(grad_output.size(0), grad_output.size(1), DType::kByte);
  auto act_input_cu = makeTransformerEngineTensor(act_input);
  auto input_cu = makeTransformerEngineTensor(grad_output);
  auto rowwise_output_cu =
      makeTransformerEngineTensor(dact_rowwise.data_ptr(), {M, N}, otype, rowwise_amax_dptr,
                                  rowwise_scale_dptr, rowwise_scale_inv_dptr);
  auto columnwise_output_cu =
      makeTransformerEngineTensor(dact_columnwise.data_ptr(), {M, N}, otype, columnwise_amax_dptr,
                                  columnwise_scale_dptr, columnwise_scale_inv_dptr);
  auto dbias_cu = makeTransformerEngineTensor(grad_bias);

  // Query workspace size and allocate workspace
  transformer_engine::TensorWrapper workspace;
  nvte_fp8_quantize_dbias_dsrelu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                    columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                    at::cuda::getCurrentCUDAStream());
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // Launch kernel
  nvte_fp8_quantize_dbias_dsrelu_x2(input_cu.data(), act_input_cu.data(), rowwise_output_cu.data(),
                                    columnwise_output_cu.data(), dbias_cu.data(), workspace.data(),
                                    at::cuda::getCurrentCUDAStream());

  return {grad_bias, dact_rowwise, dact_columnwise};
}
