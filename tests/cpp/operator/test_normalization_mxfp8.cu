#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cmath>
#include <cstring>
#include <memory>
#include <iomanip>
#include <iostream>
#include <random>

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/normalization.h>
#include <transformer_engine/transformer_engine.h>
#include "../test_common.h"

using namespace transformer_engine;
using namespace test;

namespace {

using e8m0_t = byte;

enum NormType {
  LayerNorm,
  RMSNorm
};

std::map<NormType, std::string> normToString = {
  {NormType::LayerNorm, "LayerNorm"},
  {NormType::RMSNorm, "RMSNorm"}
};

template <typename InputType, typename ScaleType, typename OutputType>
void dequantize_1x_kernel(InputType* input_ptr, ScaleType* scale_ptr, OutputType* output_ptr,
  size_t rows, size_t cols, size_t scaling_mode_x, size_t scaling_mode_y){

  const size_t block_size_Y = scaling_mode_x;   // mind the mapping Y <-- x
  const size_t block_size_X = scaling_mode_y;   //              and X <-- y
  const size_t tile_size_Y = std::max(32lu, block_size_Y);
  const size_t tile_size_X = std::max(64lu, block_size_X);
  const size_t tiles_num_Y = (rows + tile_size_Y - 1) / tile_size_Y;
  const size_t tiles_num_X = (cols + tile_size_X - 1) / tile_size_X;
  const size_t blocks_per_tile_Y = tile_size_Y / block_size_Y;
  const size_t blocks_per_tile_X = tile_size_X / block_size_X;
  const size_t blocks_per_row = (cols + block_size_X - 1) / block_size_X;

  #pragma omp parallel for proc_bind(spread) schedule(static)
  for (size_t t = 0; t < tiles_num_Y * tiles_num_X; ++t) {
      const size_t tile_Y = t / tiles_num_X;
      const size_t tile_X = t % tiles_num_X;
      const size_t tile_offset_Y = tile_Y * tile_size_Y;
      const size_t tile_offset_X = tile_X * tile_size_X;

      for (size_t ii = 0; ii < blocks_per_tile_Y; ++ii) {
          const size_t block_idx_Y = tile_Y * blocks_per_tile_Y + ii;
          const size_t block_offset_Y = ii * block_size_Y;
          const size_t i_min = tile_offset_Y + block_offset_Y;
          const size_t i_max = std::min(i_min + block_size_Y, rows);

          for (size_t jj = 0; jj < blocks_per_tile_X; ++jj) {
              const size_t block_idx_X = tile_X * blocks_per_tile_X + jj;
              const size_t block_offset_X = jj * block_size_X;
              const size_t j_min = tile_offset_X + block_offset_X;
              const size_t j_max = std::min(j_min + block_size_X, cols);

              const size_t mx_scale_idx = block_idx_Y * blocks_per_row + block_idx_X;

              // TODO: padded SFs i.e. (4,128)
              const float scale_inv = exp2f(static_cast<float>(scale_ptr[mx_scale_idx]) - FP32_EXPONENT_BIAS);
              for (size_t i = i_min; i < i_max; ++i) {
                  for (size_t j = j_min; j < j_max; ++j) {
                    const size_t idx = i * cols + j;
                    const float elem = static_cast<float>(input_ptr[idx]);
                    output_ptr[idx] = static_cast<float>(elem * scale_inv);
                  }
              }
          }
      }
  }
}

template <typename InputType, typename ScaleType>
void dequantize_2x(Tensor& input, Tensor& output_rowwise, Tensor& output_colwise)
{
  input.to_cpu();
  auto scaling_mode = input.scaling_mode();
  assert(input.shape().ndim == 2);
  assert(input.columnwise_shape().ndim == 2);

  dequantize_1x_kernel(input.cpu_dptr<InputType>(),
                       input.cpu_scale_inv_ptr<ScaleType>(),
                       output_rowwise.cpu_dptr<float>(),
                       input.shape().data[0], input.shape().data[1],
                       scaling_mode.x, scaling_mode.y);
  dequantize_1x_kernel(input.columnwise_cpu_dptr<InputType>(),
                       input.columnwise_cpu_scale_inv_ptr<ScaleType>(),
                       output_colwise.cpu_dptr<float>(),
                       input.columnwise_shape().data[0], input.columnwise_shape().data[1],
                       scaling_mode.y, scaling_mode.x);

}

template <typename InputType>
void compute_ref_stats(NormType norm_type,
                       const InputType *data, float *mu, float *rsigma,
                       const size_t N, const size_t H, const double epsilon){
  using compute_t = float;

  #pragma omp parallel for proc_bind(spread)
  for (size_t i = 0; i < N; ++i) {
    compute_t sum = 0;
    for (size_t j = 0; j < H; ++j) {
      sum += static_cast<compute_t>(data[i * H + j]);
    }
    compute_t m;
    if (norm_type == LayerNorm){
      mu[i] = sum / H;
      m = mu[i];
    } else { m = 0;}

    compute_t sum_sq = 0;
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      sum_sq += (current - m) * (current - m);
    }
    rsigma[i] = rsqrtf((sum_sq / H) + epsilon);
  }
}

template <typename InputType, typename OutputType>
void compute_ref_output(NormType norm_type,
                        const InputType *data, const InputType *gamma, const InputType *beta,
                        const float *mu, const float *rsigma,
                        const size_t N, const size_t H,
                        OutputType* output,
                        const bool zero_centered_gamma){
  using compute_t = float;

  #pragma omp parallel for proc_bind(spread)
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      compute_t g = static_cast<compute_t>(gamma[j]);
      if (zero_centered_gamma) {
        g += 1.0;
      }

      compute_t tmp;
      if (norm_type == LayerNorm) {
        tmp = (current - mu[i]) * rsigma[i] * g + static_cast<compute_t>(beta[j]);
      } else { // RMSNorm
        tmp = current * rsigma[i] * g;
      }

      output[i * H + j] = tmp;
    }
  }
}

template <typename InputType, typename OutputType>
void performTest(const size_t N, const size_t H, const bool zero_centered_gamma, NormType norm_type) {

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  const auto deviceComputeCapability = 10 * prop.major + prop.minor;
  constexpr int32_t blackwellComputeCapability = 100;
  if (deviceComputeCapability < blackwellComputeCapability) {
    GTEST_SKIP();
  }

  using WeightType = InputType;
  DType itype = TypeInfo<InputType>::dtype;
  DType wtype = TypeInfo<WeightType>::dtype;
  DType otype = TypeInfo<OutputType>::dtype;

  const std::vector<int> row_mode = {1, 32, 0};

  Tensor input({ N, H }, itype);
  Tensor z({ N, H }, otype, row_mode, true /*is_tensor_2x*/);
  Tensor gamma({ H }, wtype);
  Tensor beta({ H }, wtype);
  Tensor mu({ N }, DType::kFloat32);
  Tensor rsigma({ N }, DType::kFloat32);
  Tensor workspace;


  fillUniform(&input);
  fillUniform(&gamma);
  fillUniform(&beta);

  // Forward kernel
  float epsilon = 1e-5;
  if (norm_type == NormType::LayerNorm){
    nvte_layernorm_fwd(input.data(), gamma.data(), beta.data(), epsilon,
                       z.data(), mu.data(), rsigma.data(), workspace.data(),
                       prop.multiProcessorCount, zero_centered_gamma,
                       0);
    workspace = Tensor(workspace.shape(), workspace.dtype());
    nvte_layernorm_fwd(input.data(), gamma.data(), beta.data(), epsilon,
                       z.data(), mu.data(), rsigma.data(), workspace.data(),
                       prop.multiProcessorCount, zero_centered_gamma,
                       0);
  } else {
    nvte_rmsnorm_fwd(input.data(), gamma.data(), epsilon,
                     z.data(), rsigma.data(), workspace.data(),
                     prop.multiProcessorCount, zero_centered_gamma,
                     0);

    workspace = Tensor(workspace.shape(), workspace.dtype());
    nvte_rmsnorm_fwd(input.data(), gamma.data(), epsilon,
                     z.data(), rsigma.data(), workspace.data(),
                     prop.multiProcessorCount, zero_centered_gamma,
                     0);
  }

  Tensor dequantized_rowwise_output({ N, H }, DType::kFloat32);
  Tensor dequantized_colwise_output({ N, H }, DType::kFloat32);

  dequantize_2x<OutputType, e8m0_t>(z, dequantized_rowwise_output, dequantized_colwise_output);

  // Reference implementations
  std::unique_ptr<float[]> ref_mu = std::make_unique<float[]>(N);
  std::unique_ptr<float[]> ref_rsigma = std::make_unique<float[]>(N);
  std::unique_ptr<float[]> ref_output = std::make_unique<float[]>(N * H);


  compute_ref_stats(norm_type, input.cpu_dptr<InputType>(), ref_mu.get(),
                    ref_rsigma.get(), N, H, epsilon);
  // use the GPU stats to tighten the tolerances
  mu.to_cpu();
  rsigma.to_cpu();
  compute_ref_output(norm_type, input.cpu_dptr<InputType>(),
                     gamma.cpu_dptr<WeightType>(),
                     beta.cpu_dptr<WeightType>(),
                     mu.cpu_dptr<float>(),
                     rsigma.cpu_dptr<float>(),
                     N, H,
                     ref_output.get(),
                     zero_centered_gamma);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

  auto [atol_stats, rtol_stats] = getTolerances(DType::kFloat32);
  rtol_stats = 5e-5;
  compareResults("mu", mu, ref_mu.get(), atol_stats, rtol_stats);
  compareResults("rsigma", rsigma, ref_rsigma.get(), atol_stats, rtol_stats);

  float atol, rtol;
  if (otype == DType::kFloat8E5M2){
    atol = 1.25e-1;
    rtol = 1.25e-1;
  } else if (otype == DType::kFloat8E4M3){
    if (itype == DType::kBFloat16){
      atol = 7e-2;
      rtol = 7e-2;
    } else {
      atol = 6.25e-2;
      rtol = 6.25e-2;
    }
  }
  compareResults("output_rowwise", dequantized_rowwise_output, ref_output.get(), atol, rtol, false);
  compareResults("output_colwise", dequantized_colwise_output, ref_output.get(), atol, rtol, false);
}

std::vector<std::pair<size_t, size_t>> test_cases = {
  {32, 32},
  {128, 64},
  {768, 1024},
  {64, 2304},
  {128, 6144},
  {256, 65536},
  {2048, 12288},
};

std::vector<NormType> norms = {
  NormType::LayerNorm,
  NormType::RMSNorm
};

}  // namespace

class MxNormTestSuite : public ::testing::TestWithParam< std::tuple<NormType,
                                                                    transformer_engine::DType,
                                                                    transformer_engine::DType,
                                                                    std::pair<size_t, size_t>,
                                                                    bool>> {};

TEST_P(MxNormTestSuite, TestMxNorm) {
  using namespace transformer_engine;
  using namespace test;

  const NormType norm_type = std::get<0>(GetParam());
  const DType input_type = std::get<1>(GetParam());
  const DType output_type = std::get<2>(GetParam());
  const auto size = std::get<3>(GetParam());
  const bool zero_centered_gamma = std::get<4>(GetParam());

  TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(input_type, InputType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP8_ONLY(output_type, OutputType,
      performTest<InputType, OutputType>(size.first, size.second, zero_centered_gamma, norm_type);
    );
  );
}

INSTANTIATE_TEST_SUITE_P(
  OperatorTest,
  MxNormTestSuite,
  ::testing::Combine(
    ::testing::Values(NormType::LayerNorm, NormType::RMSNorm),
    ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
    ::testing::Values(DType::kFloat8E5M2, DType::kFloat8E4M3),
    ::testing::ValuesIn(test_cases),
    ::testing::Values(false, true)),
  [](const testing::TestParamInfo<MxNormTestSuite::ParamType>& info) {
    std::string name = normToString.at(std::get<0>(info.param)) + "_" +
      test::typeName(std::get<1>(info.param)) + "X" +
      test::typeName(std::get<2>(info.param)) + "X" +
      std::to_string(std::get<3>(info.param).first) + "X" +
      std::to_string(std::get<3>(info.param).second) + "X" +
      std::to_string(std::get<4>(info.param));
    return name;
  });
