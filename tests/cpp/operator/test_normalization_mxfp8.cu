#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cmath>
#include <cstring>
#include <memory>
#include <iomanip>
#include <iostream>
#include <random>

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/layer_norm.h>
#include <transformer_engine/rmsnorm.h>
#include <transformer_engine/transformer_engine.h>
#include "../test_common.h"

using namespace transformer_engine;
using namespace test;

namespace {

using e8m0_t = byte;

enum NormType {
  LayerNorm,
  RMSNorm
};

std::map<NormType, std::string> normToString = {
  {NormType::LayerNorm, "LayerNorm"},
  {NormType::RMSNorm, "RMSNorm"}
};

template <typename OutputType, typename ScaleType>
void dequantize(Tensor& input, Tensor& output)
{
  input.to_cpu();
  auto scaling_mode = input.scaling_mode();
  assert(input.shape().ndim == 2);
  auto nrows = input.shape().data[0];
  auto ncols = input.shape().data[1];
  auto* output_ptr = output.cpu_dptr<float>();
  const auto* input_ptr = input.cpu_dptr<OutputType>();
  const auto* scale_ptr = input.cpu_scale_inv_ptr<ScaleType>();

  const size_t n_blocks_x = (nrows + scaling_mode.x - 1) / scaling_mode.x;
  const size_t n_blocks_y = (ncols +scaling_mode.y - 1) / scaling_mode.y;

  for (size_t ii = 0; ii < n_blocks_x; ++ii) {
    const size_t i_min = ii * scaling_mode.x;
    const size_t i_max = std::min((ii + 1) * scaling_mode.x, nrows);
    for (size_t jj = 0; jj < n_blocks_y; ++jj) {
      const size_t j_min = jj * scaling_mode.y;
      const size_t j_max = std::min((jj + 1) * scaling_mode.y, ncols);
      const size_t scale_idx = ii * n_blocks_y + jj;  // TODO: padded SFs i.e. (4,128)
      float scale_inv = exp2f(static_cast<float>(scale_ptr[scale_idx]) - FP32_EXPONENT_BIAS);
      for (size_t i = i_min; i < i_max; ++i) {
        for (size_t j = j_min; j < j_max; ++j) {
          const size_t idx = i * ncols + j;
          float elem = static_cast<float>(input_ptr[idx]);
          output_ptr[idx] = static_cast<float>(elem * scale_inv);
        }
      }

    }
  }
}

template <typename InputType>
void compute_ref_stats(NormType norm_type,
                       const InputType *data, float *mu, float *rsigma,
                       const size_t N, const size_t H, const double epsilon){
  using compute_t = float;
  compute_t current, m;
  for (size_t i = 0; i < N; ++i) {
    compute_t sum = 0;
    for (size_t j = 0; j < H; ++j) {
      sum += static_cast<compute_t>(data[i * H + j]);
    }
    if (norm_type == LayerNorm){
      mu[i] = sum / H;
      m = mu[i];
    } else { m = 0;}

    compute_t sum_sq = 0;
    for (size_t j = 0; j < H; ++j) {
      current = static_cast<compute_t>(data[i * H + j]);
      sum_sq += (current - m) * (current - m);
    }
    rsigma[i] = rsqrtf((sum_sq / H) + epsilon);
  }
}

template <typename InputType, typename OutputType>
void compute_ref_output(NormType norm_type,
                        const InputType *data, const InputType *gamma, const InputType *beta,
                        const float *mu, const float *rsigma,
                        const size_t N, const size_t H,
                        OutputType* output,
                        const bool zero_centered_gamma){
  using compute_t = float;
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      compute_t g = static_cast<compute_t>(gamma[j]);
      if (zero_centered_gamma) {
        g += 1.0;
      }

      compute_t tmp;
      if (norm_type == LayerNorm) {
        tmp = (current - mu[i]) * rsigma[i] * g + static_cast<compute_t>(beta[j]);
      } else { // RMSNorm
        tmp = current * rsigma[i] * g;
      }

      output[i * H + j] = tmp;
    }
  }
}


template <typename InputType, typename OutputType>
void performTest(const size_t N, const size_t H, const bool zero_centered_gamma, NormType norm_type) {
  using WeightType = InputType;
  DType itype = TypeInfo<InputType>::dtype;
  DType wtype = TypeInfo<WeightType>::dtype;
  DType otype = TypeInfo<OutputType>::dtype;

  const std::vector<int> row_mode = {1, 32, 0};
  const std::vector<int> col_mode = {32, 1, 0};

  Tensor input({ N, H }, itype);
  Tensor z_rowwise({ N, H }, otype, row_mode);
  Tensor z_colwise({ N, H }, otype, col_mode);
  Tensor gamma({ H }, wtype);
  Tensor beta({ H }, wtype);
  Tensor mu({ N }, DType::kFloat32);
  Tensor rsigma({ N }, DType::kFloat32);
  Tensor workspace;


  fillUniform(&input);
  fillUniform(&gamma);
  fillUniform(&beta);

  // // print input tensor
  // printf("Input tensor: \n");
  // for (int i = 0; i < N; i++){
  //   for (int j = 0; j < H; j++){
  //     std::cout << "(" << i << "," << j << "): "
  //       << static_cast<float>(input.cpu_dptr<InputType>()[i * H + j])
  //       << std::endl;
  //   }
  // }


  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  // Forward kernel
  float epsilon = 1e-5;
  if (norm_type == NormType::LayerNorm){
    auto fwd_function = zero_centered_gamma ? nvte_layernorm1p_fwd_2x : nvte_layernorm_fwd_2x;
    fwd_function(input.data(), gamma.data(), beta.data(), epsilon,
                 z_rowwise.data(), z_colwise.data(), mu.data(), rsigma.data(), 0, prop.multiProcessorCount,
                 workspace.data());

    workspace = Tensor(workspace.shape(), workspace.dtype());
    fwd_function(input.data(), gamma.data(), beta.data(), epsilon,
                 z_rowwise.data(), z_colwise.data(), mu.data(), rsigma.data(), 0, prop.multiProcessorCount,
                 workspace.data());
  } else {
    auto fwd_function = zero_centered_gamma ? nvte_rmsnorm1p_fwd_2x : nvte_rmsnorm_fwd_2x;
    fwd_function(input.data(), gamma.data(), epsilon,
                 z_rowwise.data(), z_colwise.data(), rsigma.data(), 0, prop.multiProcessorCount,
                 workspace.data());

    workspace = Tensor(workspace.shape(), workspace.dtype());
    fwd_function(input.data(), gamma.data(), epsilon,
                 z_rowwise.data(), z_colwise.data(), rsigma.data(), 0, prop.multiProcessorCount,
                 workspace.data());
  }
  // z_rowwise.to_cpu();
  // for (int i = 0; i < N; i++)
  //   for (int j = 0; j < H; j++)
  //     std::cout << float(z_rowwise.cpu_dptr<OutputType>()[i * H + j]) << std::endl;

  Tensor dequantized_rowwise_output({ N, H }, DType::kFloat32);
  Tensor dequantized_colwise_output({ N, H }, DType::kFloat32);

  dequantize<OutputType, e8m0_t>(z_rowwise, dequantized_rowwise_output);
  dequantize<OutputType, e8m0_t>(z_colwise, dequantized_colwise_output);

  // Reference implementations
  std::unique_ptr<float[]> ref_mu = std::make_unique<float[]>(N);
  std::unique_ptr<float[]> ref_rsigma = std::make_unique<float[]>(N);
  std::unique_ptr<float[]> ref_output = std::make_unique<float[]>(N * H);


  compute_ref_stats(norm_type, input.cpu_dptr<InputType>(), ref_mu.get(),
                    ref_rsigma.get(), N, H, epsilon);
  // use the GPU stats to tighten the tolerances
  mu.to_cpu();
  rsigma.to_cpu();
  compute_ref_output(norm_type, input.cpu_dptr<InputType>(),
                     gamma.cpu_dptr<WeightType>(),
                     beta.cpu_dptr<WeightType>(),
                     mu.cpu_dptr<float>(),
                     rsigma.cpu_dptr<float>(),
                     N, H,
                     ref_output.get(),
                     zero_centered_gamma);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

  auto [atol_stats, rtol_stats] = getTolerances(DType::kFloat32);
  rtol_stats = 5e-5;
  compareResults("mu", mu, ref_mu.get(), atol_stats, rtol_stats);
  compareResults("rsigma", rsigma, ref_rsigma.get(), atol_stats, rtol_stats);

  float atol, rtol;
  if (otype == DType::kFloat8E5M2){
    atol = 1.25e-1;
    rtol = 1.25e-1;
  } else if (otype == DType::kFloat8E4M3){
    if (itype == DType::kBFloat16){
      atol = 6.5e-2;
      rtol = 6.5e-2;
    } else {
      atol = 6.25e-2;
      rtol = 6.25e-2;
    }
  }
  compareResults("output_rowwise", dequantized_rowwise_output, ref_output.get(), atol, rtol, false);
  compareResults("output_colwise", dequantized_colwise_output, ref_output.get(), atol, rtol, false);
}

std::vector<std::pair<size_t, size_t>> test_cases = {
  // {32, 32},
  {128, 64},
  {768, 1024},
  {64, 2304},
  {128, 6144},
  {256, 65536},
  {2048, 12288},
};

std::vector<NormType> norms = {
  NormType::LayerNorm,
  NormType::RMSNorm
};

}  // namespace

class MxNormTestSuite : public ::testing::TestWithParam< std::tuple<NormType,
transformer_engine::DType,
transformer_engine::DType,
std::pair<size_t, size_t>,
bool>> {};

TEST_P(MxNormTestSuite, TestMxNorm) {
  using namespace transformer_engine;
  using namespace test;

  const NormType norm_type = std::get<0>(GetParam());
  const DType input_type = std::get<1>(GetParam());
  const DType output_type = std::get<2>(GetParam());
  const auto size = std::get<3>(GetParam());
  const bool zero_centered_gamma = std::get<4>(GetParam());

  TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(input_type, InputType,
                                                TRANSFORMER_ENGINE_TYPE_SWITCH_FP8_ONLY(output_type, OutputType,
                                                                                        performTest<InputType, OutputType>(size.first, size.second, zero_centered_gamma, norm_type);
                                                                                        );
                                                );
}

INSTANTIATE_TEST_SUITE_P(
  OperatorTest,
  MxNormTestSuite,
  ::testing::Combine(
    ::testing::Values(NormType::LayerNorm, NormType::RMSNorm),
    ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
    ::testing::Values(DType::kFloat8E4M3, DType::kFloat8E5M2),
    ::testing::ValuesIn(test_cases),
    ::testing::Values(false, true)),
  [](const testing::TestParamInfo<MxNormTestSuite::ParamType>& info) {
    std::string name = normToString.at(std::get<0>(info.param)) + "_" +
      test::typeName(std::get<1>(info.param)) + "X" +
      test::typeName(std::get<2>(info.param)) + "X" +
      std::to_string(std::get<3>(info.param).first) + "X" +
      std::to_string(std::get<3>(info.param).second) + "X" +
      std::to_string(std::get<4>(info.param));
    return name;
  });
