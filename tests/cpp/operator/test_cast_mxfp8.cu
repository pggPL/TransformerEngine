#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cstring>
#include <iomanip>
#include <iostream>
#include <memory>
#include <random>
#include <limits>

#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/cast.h>
#include "../test_common.h"

using namespace transformer_engine;
using namespace test;

namespace {

enum ProcessingMethod {
    CAST_ONLY,
    CAST_DBIAS,
    CAST_DBIAS_DACT
};

enum dActivationType {
    Identity,
    dGeLU,
    dSiLU,
    dReLU,
    dQGeLU,
    dSReLU
};

template <typename InputType, typename OutputType, float (*OP)(const float)>
void scale_block(const ProcessingMethod processing_method,
                 const InputType* input,
                 const InputType* act_input,
                 OutputType* output_c,
                 float* dbias,
                 e8m0_t* output_scales,
                 const size_t scale_idx,
                 const size_t i_min,
                 const size_t i_max,
                 const size_t j_min,
                 const size_t j_max,
                 const size_t cols) {
    float amax = 0.0f;

    // Find the absolute maximum value in the block
    for (size_t i = i_min; i < i_max; ++i) {
        for (size_t j = j_min; j < j_max; ++j) {
            const size_t idx = i * cols + j;
            const float activation_val = OP(static_cast<float>(act_input[idx]));
            const float elt = static_cast<float>(input[idx]) * activation_val;
            dbias[j] += elt;
            if (isinf(elt) || isnan(elt)) {
                continue;
            }
            amax = std::max(amax, std::abs(elt));
        }
    }

    const e8m0_t biased_exponent = float_to_e8m0(amax * Quantized_Limits<OutputType>::max_reciprocal());
    const float scale_reciprocal = exp2f_rcp(biased_exponent);
    output_scales[scale_idx] = biased_exponent;

    // Quantize elements in the block
    for (size_t i = i_min; i < i_max; ++i) {
        for (size_t j = j_min; j < j_max; ++j) {
            const size_t idx = i * cols + j;
            const float activation_val = OP(static_cast<float>(act_input[idx]));
            const float elt = static_cast<float>(input[idx]) * activation_val;
            output_c[idx] = static_cast<OutputType>(elt * scale_reciprocal);
        }
    }
}

template <typename InputType, typename OutputType, float (*OP)(const float)>
void compute_ref_x1(const ProcessingMethod processing_method,
                    const InputType* input,
                    const InputType* act_input,
                    OutputType* output_c,
                    e8m0_t* output_scales,
                    InputType* output_dbias,
                    const size_t rows,
                    const size_t cols,
                    const size_t block_size_Y,
                    const size_t block_size_X) {
    std::vector<float> output_dbias_fp32(cols, 0);

    const size_t blocks_Y = (rows + block_size_Y - 1) / block_size_Y;
    const size_t blocks_X = (cols + block_size_X - 1) / block_size_X;

    for (size_t ii = 0; ii < blocks_Y; ++ii) {
        const size_t i_min = ii * block_size_Y;
        const size_t i_max = std::min((ii + 1) * block_size_Y, rows);
        for (size_t jj = 0; jj < blocks_X; ++jj) {
            const size_t j_min = jj * block_size_X;
            const size_t j_max = std::min((jj + 1) * block_size_X, cols);
            const size_t scale_idx = ii * blocks_X + jj;
            scale_block<InputType, OutputType, OP>(
                processing_method, input, act_input, output_c, output_dbias_fp32.data(),
                output_scales, scale_idx, i_min, i_max, j_min, j_max, cols);
        }
    }
    for (size_t j = 0; j < cols; ++j) {
        output_dbias[j] = static_cast<InputType>(output_dbias_fp32[j]);
    }
}

template <typename InputType, typename OutputType, float (*OP)(const float)>
void compute_ref_x2(const ProcessingMethod processing_method,
                    const InputType* input,
                    const InputType* act_input,
                    OutputType* output_rowwise,
                    OutputType* output_colwise,
                    e8m0_t* scales_rowwise,
                    e8m0_t* scales_colwise,
                    InputType* output_dbias,
                    const size_t rows,
                    const size_t cols,
                    const size_t block_size_Y,
                    const size_t block_size_X) {
    compute_ref_x1<InputType, OutputType, OP>(
        processing_method, input, act_input, output_rowwise, scales_rowwise, output_dbias,
        rows, cols, 1, block_size_X);
    compute_ref_x1<InputType, OutputType, OP>(
        processing_method, input, act_input, output_colwise, scales_colwise, output_dbias,
        rows, cols, block_size_Y, 1);
}

/**
 * Scaling along single dimension (either rows or columns)
 * Produces one set of output data and the corresponding data of the fused operation (dbias):
 * 1) Scaled rows + row-wise scaling factors
 *       OR
 * 2) Scaled columns + column-wise scaling factors
 */

template <typename InputType, typename OutputType, float (*OP)(const float)>
void performTest_x1(const ProcessingMethod processing_method,
                    const size_t rows,
                    const size_t cols,
                    const size_t block_size_rows,
                    const size_t block_size_cols,
                    InputsFillCase fill_case) {
    using namespace test;
    using EncodingType = fp32;
    DType itype = TypeInfo<InputType>::dtype;
    DType otype = TypeInfo<OutputType>::dtype;

    const size_t blocks_Y = (rows + block_size_rows - 1) / block_size_rows;
    const size_t blocks_X = (cols + block_size_cols - 1) / block_size_cols;
    const size_t blocks_num = blocks_Y * blocks_X;

    const int block_rows_dim = static_cast<int>(block_size_rows);
    const int block_cols_dim = static_cast<int>(block_size_cols);
    const int is_delayed_scaling = false;

    Tensor input({ rows, cols }, itype);
    Tensor act_input({ rows, cols }, itype);
    Tensor output_c({ rows, cols }, otype, { block_rows_dim, block_cols_dim, is_delayed_scaling});
    Tensor output_dbias({ cols }, itype);

    std::unique_ptr<OutputType[]> ref_output_c = std::make_unique<OutputType[]>(rows * cols);
    std::unique_ptr<InputType[]> ref_output_dbias = std::make_unique<InputType[]>(cols);
    std::unique_ptr<e8m0_t[]> ref_output_scales = std::make_unique<e8m0_t[]>(blocks_Y * blocks_X);

    fillCase<EncodingType>(&input, fill_case);
    fillUniform(&act_input);

    Tensor workspace;
    switch (processing_method) {
        case ProcessingMethod::CAST_ONLY: {
            nvte_fp8_quantize(input.data(), output_c.data(), 0);
            workspace = Tensor(workspace.shape(), workspace.dtype());

            nvte_fp8_quantize(input.data(), output_c.data(), 0);
            break;
        }
        case ProcessingMethod::CAST_DBIAS: {
            nvte_fp8_quantize_dbias(input.data(),
                                    output_c.data(),
                                    output_dbias.data(),
                                    workspace.data(),
                                    0);
            workspace = Tensor(workspace.shape(), workspace.dtype());

            nvte_fp8_quantize_dbias(input.data(),
                                    output_c.data(),
                                    output_dbias.data(),
                                    workspace.data(),
                                    0);
            break;
        }
        case ProcessingMethod::CAST_DBIAS_DACT: {
            nvte_fp8_quantize_dbias_dgelu(input.data(),
                                          act_input.data(),
                                          output_c.data(),
                                          output_dbias.data(),
                                          workspace.data(),
                                          0);
            workspace = Tensor(workspace.shape(), workspace.dtype());

            nvte_fp8_quantize_dbias_dgelu(input.data(),
                                          act_input.data(),
                                          output_c.data(),
                                          output_dbias.data(),
                                          workspace.data(),
                                          0);
            break;
        }
    }

    hipDeviceSynchronize();
    auto err = hipGetLastError();
    ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

    compute_ref_x1<InputType, OutputType, OP>(processing_method,
                                              input.cpu_dptr<InputType>(),
                                              act_input.cpu_dptr<InputType>(),
                                              ref_output_c.get(),
                                              ref_output_scales.get(),
                                              ref_output_dbias.get(),
                                              rows,
                                              cols,
                                              block_size_rows,
                                              block_size_cols);

    auto [atol, rtol] = getTolerances(otype);
    compareResults("output_c", output_c, ref_output_c.get(), atol, rtol);
    compare_e8m0_scaling_factors("scales", output_c.cpu_scale_inv_ptr<e8m0_t>(), ref_output_scales.get(), blocks_num);

    if (processing_method == ProcessingMethod::CAST_DBIAS || processing_method == ProcessingMethod::CAST_DBIAS_DACT) {
        auto [atol_dbias, rtol_dbias] = getTolerances(itype);
        rtol_dbias *= 4;
        if (itype == DType::kFloat32) {
            atol_dbias = 1e-4;
        }
        compareResults("output_dbias", output_dbias, ref_output_dbias.get(), atol_dbias, rtol_dbias);
    }
}

/**
 * Scaling along both dimensions (rows and columns)
 * Produces two sets of scaled output data and the corresponding data of the fused operation (dbias):
 * 1) Scaled rows + row-wise scaling factors
 *      AND
 * 2) Scaled columns + column-wise scaling factors
 */
template <typename InputType, typename OutputType, float (*OP)(const float)>
void performTest_x2(const ProcessingMethod processing_method,
                    const size_t rows,
                    const size_t cols,
                    const size_t block_size_rows,
                    const size_t block_size_cols,
                    InputsFillCase fill_case) {
    using namespace test;
    using EncodingType = fp32;
    DType itype = TypeInfo<InputType>::dtype;
    DType otype = TypeInfo<OutputType>::dtype;

    const size_t blocks_Y = (rows + block_size_rows - 1) / block_size_rows;
    const size_t blocks_X = (cols + block_size_cols - 1) / block_size_cols;
    const size_t blocks_num_rowwise = rows * blocks_X;
    const size_t blocks_num_colwise = blocks_Y * cols;

    const int block_rows_dim = static_cast<int>(block_size_rows);
    const int block_cols_dim = static_cast<int>(block_size_cols);
    const int is_delayed_scaling = false;

    Tensor input({ rows, cols }, itype);
    Tensor act_input({ rows, cols }, itype);
    Tensor output_rowwise({ rows, cols }, otype, { 1, block_cols_dim, is_delayed_scaling});
    Tensor output_colwise({ rows, cols }, otype, { block_rows_dim, 1, is_delayed_scaling});
    Tensor output_dbias({ cols }, itype);

    std::unique_ptr<OutputType[]> ref_output_c_rowwise = std::make_unique<OutputType[]>(rows * cols);
    std::unique_ptr<OutputType[]> ref_output_c_colwise = std::make_unique<OutputType[]>(rows * cols);
    std::unique_ptr<e8m0_t[]> ref_scales_rowwise = std::make_unique<e8m0_t[]>(rows * blocks_X);
    std::unique_ptr<e8m0_t[]> ref_scales_colwise = std::make_unique<e8m0_t[]>(blocks_Y * cols);
    std::unique_ptr<InputType[]> ref_output_dbias = std::make_unique<InputType[]>(cols);

    fillCase<EncodingType>(&input, fill_case);
    fillUniform(&act_input);

    Tensor workspace;
    switch (processing_method) {
        case ProcessingMethod::CAST_ONLY: {
            nvte_fp8_quantize_x2(input.data(), output_rowwise.data(), output_colwise.data(), 0);
            workspace = Tensor(workspace.shape(), workspace.dtype());

            nvte_fp8_quantize_x2(input.data(), output_rowwise.data(), output_colwise.data(), 0);
            break;
        }
        case ProcessingMethod::CAST_DBIAS: {
            nvte_fp8_quantize_dbias_x2(input.data(),
                                       output_rowwise.data(),
                                       output_colwise.data(),
                                       output_dbias.data(),
                                       workspace.data(),
                                       0);
            workspace = Tensor(workspace.shape(), workspace.dtype());

            nvte_fp8_quantize_dbias_x2(input.data(),
                                       output_rowwise.data(),
                                       output_colwise.data(),
                                       output_dbias.data(),
                                       workspace.data(),
                                       0);
            break;
        }
        case ProcessingMethod::CAST_DBIAS_DACT: {
            nvte_fp8_quantize_dbias_dgelu_x2(input.data(),
                                             act_input.data(),
                                             output_rowwise.data(),
                                             output_colwise.data(),
                                             output_dbias.data(),
                                             workspace.data(),
                                             0);
            workspace = Tensor(workspace.shape(), workspace.dtype());

            nvte_fp8_quantize_dbias_dgelu_x2(input.data(),
                                             act_input.data(),
                                             output_rowwise.data(),
                                             output_colwise.data(),
                                             output_dbias.data(),
                                             workspace.data(),
                                             0);
            break;
        }
    }

    hipDeviceSynchronize();
    auto err = hipGetLastError();
    ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

    compute_ref_x2<InputType, OutputType, OP>(processing_method,
                                              input.cpu_dptr<InputType>(),
                                              act_input.cpu_dptr<InputType>(),
                                              ref_output_c_rowwise.get(),
                                              ref_output_c_colwise.get(),
                                              ref_scales_rowwise.get(),
                                              ref_scales_colwise.get(),
                                              ref_output_dbias.get(),
                                              rows,
                                              cols,
                                              block_size_rows,
                                              block_size_cols);

    auto [atol, rtol] = getTolerances(otype);
    compareResults("output_c_rowwise", output_rowwise, ref_output_c_rowwise.get(), atol, rtol);
    compareResults("output_c_colwise", output_colwise, ref_output_c_colwise.get(), atol, rtol);
    compare_e8m0_scaling_factors("scales_rowwise", output_rowwise.cpu_scale_inv_ptr<e8m0_t>(),
                                 ref_scales_rowwise.get(), blocks_num_rowwise);
    compare_e8m0_scaling_factors("scales_colwise", output_colwise.cpu_scale_inv_ptr<e8m0_t>(),
                                 ref_scales_colwise.get(), blocks_num_colwise);

    if (processing_method == ProcessingMethod::CAST_DBIAS || processing_method == ProcessingMethod::CAST_DBIAS_DACT) {
        auto [atol_dbias, rtol_dbias] = getTolerances(itype);
        rtol_dbias *= 4;
        if (itype == DType::kFloat32) {
            atol_dbias = 1e-4;
        }
        compareResults("output_dbias", output_dbias, ref_output_dbias.get(), atol_dbias, rtol_dbias);
    }
}

std::vector<std::pair<size_t, size_t>> matrix_sizes = {
    {128, 128},
    {256, 256},
    {768, 1024},
    // {256, 65536},
    // {2048, 12288},
    // {65536, 128},
    // {16384, 6144},
};

std::vector<std::pair<size_t, size_t>> block_sizes = {
    {1, 32},
    {32, 1},
    {32, 32},
};

std::vector<InputsFillCase> input_scenarios = {
    InputsFillCase::uniform,
    // InputsFillCase::zeros,
    // InputsFillCase::zero_to_minNorm,
    // InputsFillCase::minNorm_to_maxNorm,
    // InputsFillCase::maxNorm_to_inf
};

std::vector<ProcessingMethod> processing_methods = {
    ProcessingMethod::CAST_ONLY,
    ProcessingMethod::CAST_DBIAS,
    ProcessingMethod::CAST_DBIAS_DACT,
};

// Only dGeLU activation tests are supported
std::vector<dActivationType> dActivation_types = {
    dActivationType::Identity,
    dActivationType::dGeLU,
    // dActivationType::dSiLU,
    // dActivationType::dReLU,
    // dActivationType::dQGeLU,
    // dActivationType::dSReLU,
};

}  // namespace

class FusedCastMXFP8TestSuite : public ::testing::TestWithParam
    <std::tuple<ProcessingMethod,
                dActivationType,
                std::pair<size_t, size_t>,
                std::pair<size_t, size_t>,
                transformer_engine::DType,
                transformer_engine::DType,
                InputsFillCase>> {};

#define DACT_FUNC_SWITCH(OP_FUNC_TYPE, OP, ...) \
switch (OP_FUNC_TYPE) { \
    case dActivationType::Identity: { constexpr auto OP = &identity; { __VA_ARGS__ } } break; \
    case dActivationType::dGeLU:    { constexpr auto OP = &dgelu;    { __VA_ARGS__ } } break; \
    case dActivationType::dSiLU:    { constexpr auto OP = &dsilu;    { __VA_ARGS__ } } break; \
    case dActivationType::dReLU:    { constexpr auto OP = &drelu;    { __VA_ARGS__ } } break; \
    case dActivationType::dQGeLU:   { constexpr auto OP = &dqgelu;   { __VA_ARGS__ } } break; \
    case dActivationType::dSReLU:   { constexpr auto OP = &dsrelu;   { __VA_ARGS__ } } break; \
}

TEST_P(FusedCastMXFP8TestSuite, TestFusedCastMXFP8) {
    // Skip tests for pre-Blackwell architectures
    if (getDeviceComputeCapability() < blackwellComputeCapability) {
        GTEST_SKIP();
    }

    using namespace transformer_engine;
    using namespace test;

    const ProcessingMethod processing_method = std::get<0>(GetParam());
    const dActivationType dAct_type = std::get<1>(GetParam());
    const auto matrix_size = std::get<2>(GetParam());
    const auto block_size = std::get<3>(GetParam());
    const DType input_type = std::get<4>(GetParam());
    const DType output_type = std::get<5>(GetParam());
    const InputsFillCase fill_case = std::get<6>(GetParam());

    // Skips non dAct tests if the dActivation type is not an identity
    if (processing_method != ProcessingMethod::CAST_DBIAS_DACT
        && dAct_type != dActivationType::Identity) {
        GTEST_SKIP();
    }

    // Skips dAct tests if the dActivation type is an identity
    if (processing_method == ProcessingMethod::CAST_DBIAS_DACT
        && dAct_type == dActivationType::Identity) {
        GTEST_SKIP();
    }

    DACT_FUNC_SWITCH(dAct_type, OP,
        TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(input_type, InputType,
            TRANSFORMER_ENGINE_TYPE_SWITCH_FP8_ONLY(output_type, OutputType,
                if (block_size.first == 1 || block_size.second == 1) {
                    performTest_x1<InputType, OutputType, OP>(
                        processing_method, matrix_size.first, matrix_size.second,
                        block_size.first, block_size.second, fill_case);
                } else {
                    performTest_x2<InputType, OutputType, OP>(
                        processing_method, matrix_size.first, matrix_size.second,
                        block_size.first, block_size.second, fill_case);
                }
            );
        );
    );
}

std::string to_string(const ProcessingMethod method) {
    switch (method) {
        case ProcessingMethod::CAST_ONLY:       return "CAST_ONLY";
        case ProcessingMethod::CAST_DBIAS:      return "CAST_DBIAS";
        case ProcessingMethod::CAST_DBIAS_DACT: return "CAST_DBIAS_DACT";
        default: return "";
    }
}

std::string to_string(const dActivationType dAct_type) {
    switch (dAct_type) {
        case dActivationType::Identity: return "Identity";
        case dActivationType::dGeLU:    return "dGeLU";
        case dActivationType::dSiLU:    return "dSiLU";
        case dActivationType::dReLU:    return "dReLU";
        case dActivationType::dQGeLU:   return "dQGeLU";
        case dActivationType::dSReLU:   return "dSReLU";
        default: return "";
    }
}

INSTANTIATE_TEST_SUITE_P(
    OperatorTest,
    FusedCastMXFP8TestSuite,
    ::testing::Combine(
        ::testing::ValuesIn(processing_methods),
        ::testing::ValuesIn(dActivation_types),
        ::testing::ValuesIn(matrix_sizes),
        ::testing::ValuesIn(block_sizes),
        ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
        ::testing::Values(DType::kFloat8E4M3, DType::kFloat8E5M2),
        ::testing::ValuesIn(input_scenarios)),
    [](const testing::TestParamInfo<FusedCastMXFP8TestSuite::ParamType>& info) {
        std::string name = to_string(std::get<0>(info.param)) + "X" +
                           to_string(std::get<1>(info.param)) + "X" +
                           std::to_string(std::get<2>(info.param).first) + "X" +
                           std::to_string(std::get<2>(info.param).second) + "X" +
                           std::to_string(std::get<3>(info.param).first) + "X" +
                           std::to_string(std::get<3>(info.param).second) + "X" +
                           test::typeName(std::get<4>(info.param)) + "X" +
                           test::typeName(std::get<5>(info.param)) + "X" +
                           test::caseName(std::get<6>(info.param));
        return name;
    });
