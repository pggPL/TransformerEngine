#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cstring>
#include <iomanip>
#include <iostream>
#include <memory>
#include <random>
#include <limits>

#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/cast.h>
#include "../test_common.h"

using namespace transformer_engine;
using namespace test;

namespace {

constexpr int exponent_bias = 127;
// constexpr bool saturated = true; 

template <typename OType, typename CType>
int compute_shared_unbiased_exponent(const CType amax) {
    if (amax == 0.0f) {
        return 0;
    }
    const int exponent = floorf(log2f(amax))
                         - floorf(log2f(Quantized_Limits<OType>::max()));
    
    const int exponent_clamped = (exponent < -127) ? -127 : exponent;
    return exponent_clamped;
}

template <typename OType, typename CType>
CType clamp_number(CType elt, CType scale) {
    const CType elt_scaled = elt * scale;

    // // In accordance with the OCP MX specification
    // // Infs and NaNs are not clamped
    // if (isinf(elt_scaled) || isnan(elt_scaled)) {
    //     return elt_scaled;
    // }
    // // P_{i} is set to zero if the corresponding input V_{i} is a subnormal Float32 number
    // if (elt_scaled < static_cast<CType>(Numeric_Traits<float>::minNorm)) {
    //     return 0.0f;
    // }
    // // when quantizing V_{i}/X, normal numbers that exceed the representable range of the
    // // element format are clamped to the maximum representable value, preserving the sign
    // if (elt_scaled > static_cast<CType>(Numeric_Traits<OType>::maxNorm)) {
    //     return static_cast<CType>(Numeric_Traits<OType>::maxNorm);
    // }
    return elt_scaled;
}

template <typename InputType, typename OutputType>
void process_block(const InputType* data,
                   OutputType* output_c,
                   byte* output_scales,
                   const size_t scale_idx,
                   const size_t i_min,
                   const size_t i_max,
                   const size_t j_min,
                   const size_t j_max,
                   const size_t cols) {
    using ComputeType = float;
    ComputeType amax = 0.0f;

    // Find the absolute maximum value in the block
    for (size_t i = i_min; i < i_max; ++i) {
        for (size_t j = j_min; j < j_max; ++j) {
            const size_t idx = i * cols + j;
            const ComputeType elt = static_cast<ComputeType>(data[idx]);
            if (isinf(elt) || isnan(elt)) {
                continue;
            }
            amax = std::max(amax, std::abs(elt));
        }
    }

    const int unbiased_exponent = compute_shared_unbiased_exponent<OutputType>(amax);
    output_scales[scale_idx] = static_cast<byte>(unbiased_exponent + exponent_bias);
    const ComputeType scale_reciprocal = powf(2.0f, -unbiased_exponent);

    // Quantize elements in the block
    for (size_t i = i_min; i < i_max; ++i) {
        for (size_t j = j_min; j < j_max; ++j) {
            const size_t idx = i * cols + j;
            const ComputeType elt = static_cast<ComputeType>(data[idx]);
            output_c[idx] = static_cast<OutputType>(elt * scale_reciprocal);
        }
    }
}

template <typename InputType, typename OutputType>
void compute_ref(const InputType* data,
                 OutputType* output_c,
                 byte* output_scales,
                 const size_t rows,
                 const size_t cols,
                 const size_t block_size_Y,
                 const size_t block_size_X) {
    using ComputeType = float;

    const size_t blocks_Y = (rows + block_size_Y - 1) / block_size_Y;
    const size_t blocks_X = (cols + block_size_X - 1) / block_size_X;

    for (size_t ii = 0; ii < blocks_Y; ++ii) {
        const size_t i_min = ii * block_size_Y;
        const size_t i_max = std::min((ii + 1) * block_size_Y, rows);
        for (size_t jj = 0; jj < blocks_X; ++jj) {
            const size_t j_min = jj * block_size_X;
            const size_t j_max = std::min((jj + 1) * block_size_X, cols);
            const size_t scale_idx = ii * blocks_X + jj;
            process_block(data, output_c, output_scales, scale_idx, i_min, i_max, j_min, j_max, cols);
        }
    }
}

template <typename InputType, typename CastToType>
void print_data(const std::string& data_name,
                const InputType* data,
                const size_t rows,
                const size_t cols,
                const bool shiftBias = false) {
    std::cout << data_name << std::endl;
    for (size_t i = 0; i < rows; ++i) {
        for (size_t j = 0; j < cols; ++j) {
            const size_t idx = i * cols + j;
            if (shiftBias) {
                std::cout << static_cast<CastToType>(data[idx]) - exponent_bias << "  ";
            } else {
                std::cout << static_cast<CastToType>(data[idx]) << "  ";
            }
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}


template <typename InputType, typename OutputType>
void print_all_data(const InputType* data,
                    const OutputType* output_c,
                    const byte* output_scales,
                    const size_t rows,
                    const size_t cols,
                    const size_t blocks_Y,
                    const size_t blocks_X) {
    print_data<InputType, float>("Input", data, rows, cols);
    print_data<OutputType, float>("Output", output_c, rows, cols);
    print_data<byte, int>("Shared exponents (biased)", output_scales, blocks_Y, blocks_X);
    print_data<byte, int>("Shared exponents", output_scales, blocks_Y, blocks_X, true);
}

template <typename InputType, typename OutputType>
void performTest(const size_t rows,
                 const size_t cols,
                 const size_t block_size_rows,
                 const size_t block_size_cols,
                 InputsFillCase fill_case) {
    using namespace test;
    using EncodingType = fp32;
    DType itype = TypeInfo<InputType>::dtype;
    DType otype = TypeInfo<OutputType>::dtype;
    DType scale_type = TypeInfo<byte>::dtype;

    const size_t blocks_Y = (rows + block_size_rows - 1) / block_size_rows;
    const size_t blocks_X = (cols + block_size_cols - 1) / block_size_cols;

    Tensor input({ rows, cols }, itype);
    Tensor output_c({ rows, cols }, otype);
    Tensor output_scales({ blocks_Y, blocks_X }, scale_type);

    std::unique_ptr<OutputType[]> ref_output_c = std::make_unique<OutputType[]>(rows * cols);
    std::unique_ptr<byte[]> ref_output_scales = std::make_unique<byte[]>(blocks_Y * blocks_X);

    // fillUniform(&input);
    fillCase<EncodingType>(&input, fill_case);

    Tensor workplace_scales;
    nvte_cast_mxfp8(input.data(), output_c.data(), workplace_scales.data(), 0);

    workplace_scales = Tensor(workplace_scales.shape(), workplace_scales.dtype());

    constexpr bool warm_up = true;
    if (warm_up) {
        constexpr int iterations = 2;
        for (int i = 0; i < iterations; ++i) {
            nvte_cast_mxfp8(input.data(), output_c.data(), workplace_scales.data(), 0);
        }
    }
    nvte_cast_mxfp8(input.data(), output_c.data(), workplace_scales.data(), 0);

    hipDeviceSynchronize();
    auto err = hipGetLastError();
    ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

    compute_ref<InputType, OutputType>(input.cpu_dptr<InputType>(),
                                       ref_output_c.get(),
                                       ref_output_scales.get(),
                                       rows,
                                       cols,
                                       block_size_rows,
                                       block_size_cols);

    constexpr bool print_data = false;
                                    
    if (print_data) {
        std::cout << "\t ===== CPU ===== \n" << std::endl; 
        print_all_data<InputType, OutputType>(input.cpu_dptr<InputType>(),
                                              ref_output_c.get(),
                                              ref_output_scales.get(),
                                              rows,
                                              cols,
                                              blocks_Y,
                                              blocks_X);

        std::cout << "\t ===== GPU ===== \n" << std::endl; 
        input.to_cpu();
        output_c.to_cpu();
        workplace_scales.to_cpu();
        print_all_data<InputType, OutputType>(input.cpu_dptr<InputType>(),
                                              output_c.cpu_dptr<OutputType>(),
                                              workplace_scales.cpu_dptr<byte>(),
                                              rows,
                                              cols,
                                              blocks_Y,
                                              blocks_X);
    }

    // if (isFp8Type(otype)) {
    //     auto [atol_amax, rtol_amax] = getTolerances(DType::kFloat32);
    //     compareResults("amax", output_t.amax(), ref_amax, atol_amax, rtol_amax);
    // }
    auto [atol, rtol] = getTolerances(otype);
    compareResults("output_c", output_c, ref_output_c.get(), atol, rtol);
    compareResults("scales", workplace_scales, ref_output_scales.get(), atol, rtol);
    // compareResults("output_t", output_t, ref_output_t.get(), atol, rtol);
}

std::vector<std::pair<size_t, size_t>> matrix_sizes = {
    // {8, 8},
    // {32, 32},
    // {64, 64},
    // {256, 256},
    // {768, 1024},
    // {256, 65536},
    // {2048, 12288},
    // {65536, 128},
    {16384, 6144},
};

std::vector<std::pair<size_t, size_t>> block_sizes = {
    {1, 32},
    // {1, 64},
    // {1, 128},
    // {32, 32},
    // {64, 64},
    // {128, 128}
};

// std::vector<std::pair<size_t, size_t>> matrix_size = {
//     {4096, 3072},
//     {4096, 4096},
//     {4096, 5440},
//     {16384, 1024},
//     {16384, 3072},
//     {16384, 6144},
// };
}  // namespace

std::vector<InputsFillCase> input_scenarios = {
    InputsFillCase::uniform,
    // InputsFillCase::zeros,
    // InputsFillCase::zero_to_minNorm,
    // InputsFillCase::minNorm_to_maxNorm,
    // InputsFillCase::maxNorm_to_inf
};

class CastMXFP8TestSuite : public ::testing::TestWithParam<std::tuple<std::pair<size_t, size_t>,
                                                                      std::pair<size_t, size_t>,
                                                                      transformer_engine::DType,
                                                                      transformer_engine::DType,
                                                                      InputsFillCase>> {};

TEST_P(CastMXFP8TestSuite, TestCastMXFP8) {
    using namespace transformer_engine;
    using namespace test;

    const auto matrix_size = std::get<0>(GetParam());
    const auto block_size = std::get<1>(GetParam());
    const DType input_type = std::get<2>(GetParam());
    const DType output_type = std::get<3>(GetParam());
    const InputsFillCase fill_case = std::get<4>(GetParam());

    TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(input_type, InputType,
        TRANSFORMER_ENGINE_TYPE_SWITCH_FP8_ONLY(output_type, OutputType,
            performTest<InputType, OutputType>(matrix_size.first, matrix_size.second,
                                               block_size.first, block_size.second, fill_case);
        );
    );
}


INSTANTIATE_TEST_SUITE_P(
    OperatorTest,
    CastMXFP8TestSuite,
    ::testing::Combine(
        ::testing::ValuesIn(matrix_sizes),
        ::testing::ValuesIn(block_sizes),
        // ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
        ::testing::Values(DType::kBFloat16),
        // ::testing::Values(DType::kFloat8E4M3, DType::kFloat8E5M2),
        ::testing::Values(DType::kFloat8E4M3),
        ::testing::ValuesIn(input_scenarios)),
    [](const testing::TestParamInfo<CastMXFP8TestSuite::ParamType>& info) {
        std::string name = std::to_string(std::get<0>(info.param).first) + "X" +
                           std::to_string(std::get<0>(info.param).second) + "X" +
                           std::to_string(std::get<1>(info.param).first) + "X" +
                           std::to_string(std::get<1>(info.param).second) + "X" +
                           test::typeName(std::get<2>(info.param)) + "X" +
                           test::typeName(std::get<3>(info.param)) + "X" +
                           test::caseName(std::get<4>(info.param));
        return name;
    });
